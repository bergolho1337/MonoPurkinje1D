#include "hip/hip_runtime.h"
#include "beeler_reuter_1977.h"
#include <stddef.h>
#include <stdint.h>
#include "model_gpu_utils.h"

extern "C" SET_ODE_INITIAL_CONDITIONS_GPU(set_model_initial_conditions_gpu) 
{

    print_to_stdout_and_file("Using beller_reuter_1977 GPU model\n");

    // execution configuration
    const int GRID  = (num_volumes + BLOCK_SIZE - 1)/BLOCK_SIZE;

    size_t size = num_volumes*sizeof(real);

    check_cuda_error(hipMallocPitch((void **) &(*sv), &pitch_h, size, (size_t )NEQ));
    check_cuda_error(hipMemcpyToSymbol(HIP_SYMBOL(pitch), &pitch_h, sizeof(size_t)));


    kernel_set_model_inital_conditions <<<GRID, BLOCK_SIZE>>>(*sv, num_volumes);

    check_cuda_error( hipPeekAtLastError() );
    hipDeviceSynchronize();
    return pitch_h;

}

extern "C" SOLVE_MODEL_ODES_GPU(solve_model_odes_gpu) {

    // execution configuration
    const int GRID  = ((int)num_cells_to_solve + BLOCK_SIZE - 1)/BLOCK_SIZE;


    size_t stim_currents_size = sizeof(real)*num_cells_to_solve;
    size_t cells_to_solve_size = sizeof(uint32_t)*num_cells_to_solve;

    real *stims_currents_device;
    check_cuda_error(hipMalloc((void **) &stims_currents_device, stim_currents_size));
    check_cuda_error(hipMemcpy(stims_currents_device, stim_currents, stim_currents_size, hipMemcpyHostToDevice));


    //the array cells to solve is passed when we are using and adapative mesh
    uint32_t *cells_to_solve_device = NULL;
    if(cells_to_solve != NULL) {
        check_cuda_error(hipMalloc((void **) &cells_to_solve_device, cells_to_solve_size));
        check_cuda_error(hipMemcpy(cells_to_solve_device, cells_to_solve, cells_to_solve_size, hipMemcpyHostToDevice));
    }
    solve_gpu <<<GRID, BLOCK_SIZE>>>(dt, sv, stims_currents_device, cells_to_solve_device, num_cells_to_solve, num_steps);

    check_cuda_error( hipPeekAtLastError() );

    check_cuda_error(hipFree(stims_currents_device));
    if(cells_to_solve_device) check_cuda_error(hipFree(cells_to_solve_device));

}

__global__ void kernel_set_model_inital_conditions(real *sv, int num_volumes) {
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;

    if (threadID < num_volumes) {

        *((real * )((char *) sv + pitch * 0) + threadID) = -84.624;        // V
        *((real * )((char *) sv + pitch * 1) + threadID) = 0.011;          // m
        *((real * )((char *) sv + pitch * 2) + threadID) = 0.988;          // h
        *((real * )((char *) sv + pitch * 3) + threadID) = 0.975;          // j
        *((real * )((char *) sv + pitch * 4) + threadID) = 1e-4;           // Cai
        *((real * )((char *) sv + pitch * 5) + threadID) = 0.003;          // d
        *((real * )((char *) sv + pitch * 6) + threadID) = 0.994;          // f
        *((real * )((char *) sv + pitch * 7) + threadID) = 0.0001;         // x1
    }
}

// Solving the model for each cell in the tissue matrix ni x nj
__global__ void solve_gpu(real dt, real *sv, real* stim_currents,
                          uint32_t *cells_to_solve, uint32_t num_cells_to_solve,
                          int num_steps)
{
    int threadID = blockDim.x * blockIdx.x + threadIdx.x;
    int sv_id;

    // Each thread solves one cell model
    if(threadID < num_cells_to_solve) {
        if(cells_to_solve)
            sv_id = cells_to_solve[threadID];
        else
            sv_id = threadID;

        real rDY[NEQ];

        for (int n = 0; n < num_steps; ++n) {

            RHS_gpu(sv, rDY, stim_currents[threadID], sv_id);

            for(int i = 0; i < NEQ; i++) {
                *((real *) ((char *) sv + pitch * i) + sv_id) = dt * rDY[i] + *((real *) ((char *) sv + pitch * i) + sv_id);
            }            

        }

    }
}

inline __device__ void RHS_gpu(real *sv_, real *rDY_, real stim_current, int threadID_) {

    // State variables
    const real V_old_ = *((real*)((char*)sv_ + pitch * 0) + threadID_);
    const real m_old_ = *((real*)((char*)sv_ + pitch * 1) + threadID_);
    const real h_old_ = *((real*)((char*)sv_ + pitch * 2) + threadID_);
    const real j_old_ = *((real*)((char*)sv_ + pitch * 3) + threadID_);
    const real Cai_old_ = *((real*)((char*)sv_ + pitch * 4) + threadID_);
    const real d_old_ = *((real*)((char*)sv_ + pitch * 5) + threadID_);
    const real f_old_ = *((real*)((char*)sv_ + pitch * 6) + threadID_);
    const real x1_old_ = *((real*)((char*)sv_ + pitch * 7) + threadID_);

    // Constants
    const real C = 0.01;
    const real g_na = 4e-2;
    const real E_na = 50;
    const real g_nac = 3e-5;
    const real g_s = 9e-4;

    // Algebraics
    real alpha_m = ( - 1.00000*(V_old_+47.0000))/(exp( - 0.100000*(V_old_+47.0000)) - 1.00000);
    real beta_m =  40.0000*exp( - 0.0560000*(V_old_+72.0000));
    real alpha_h =  0.126000*exp( - 0.250000*(V_old_+77.0000));
    real beta_h = 1.70000/(exp( - 0.0820000*(V_old_+22.5000))+1.00000);
    real alpha_j = ( 0.0550000*exp( - 0.250000*(V_old_+78.0000)))/(exp( - 0.200000*(V_old_+78.0000))+1.00000);
    real beta_j = 0.300000/(exp( - 0.100000*(V_old_+32.0000))+1.00000);
    real alpha_d = ( 0.0950000*exp(- (V_old_ - 5.00000)/100.000))/(1.00000+exp(- (V_old_ - 5.00000)/13.8900));
    real beta_d = ( 0.0700000*exp(- (V_old_+44.0000)/59.0000))/(1.00000+exp((V_old_+44.0000)/20.0000));
    real alpha_f = ( 0.0120000*exp(- (V_old_+28.0000)/125.000))/(1.00000+exp((V_old_+28.0000)/6.67000));
    real beta_f = ( 0.00650000*exp(- (V_old_+30.0000)/50.0000))/(1.00000+exp(- (V_old_+30.0000)/5.00000));
    real alpha_x1 = ( 0.000500000*exp((V_old_+50.0000)/12.1000))/(1.00000+exp((V_old_+50.0000)/17.5000));
    real beta_x1 = ( 0.00130000*exp(- (V_old_+20.0000)/16.6700))/(1.00000+exp(- (V_old_+20.0000)/25.0000));
    real E_s = - 82.3000 -  13.0287*log( Cai_old_*0.00100000);
    real i_s =  g_s*d_old_*f_old_*(V_old_ - E_s);
    real i_na =  ( g_na*powf(m_old_, 3.00000)*h_old_*j_old_+g_nac)*(V_old_ - E_na);
    real i_x1 = ( x1_old_*0.00800000*(exp( 0.0400000*(V_old_+77.0000)) - 1.00000))/exp( 0.0400000*(V_old_+35.0000));
    real i_k1 =  0.00350000*(( 4.00000*(exp( 0.0400000*(V_old_+85.0000)) - 1.00000))/(exp( 0.0800000*(V_old_+53.0000))+exp( 0.0400000*(V_old_+53.0000)))+( 0.200000*(V_old_+23.0000))/(1.00000 - exp( - 0.0400000*(V_old_+23.0000))));
    real i_stim = stim_current;

    // Rates
    rDY_[0] = (i_stim - (i_na+i_s+i_x1+i_k1))/C;
    rDY_[1] = alpha_m*(1.00000 - m_old_) -  beta_m*m_old_;
    rDY_[2] = alpha_h*(1.00000 - h_old_) -  beta_h*h_old_;
    rDY_[3] = alpha_j*(1.00000 - j_old_) -  beta_j*j_old_;
    rDY_[4] = ( - 0.0100000*i_s)/1.00000+ 0.0700000*(0.000100000 - Cai_old_);
    rDY_[5] = alpha_d*(1.00000 - d_old_) -  beta_d*d_old_;
    rDY_[6] = alpha_f*(1.00000 - f_old_) -  beta_f*f_old_;
    rDY_[7] = alpha_x1*(1.00000 - x1_old_) -  beta_x1*x1_old_;

}
